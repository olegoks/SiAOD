
#include <hip/hip_runtime.h>
//#include "MemoryManagerGPU.h"
//#include "cuda_runtime_api.h"
//
//void MemoryManagerGPU::AllocateMemoryForPrimitives() {
//
//	Vertex3D* device_vertexs;
//	Normal3D* device_normals;
//	Polygon3D* device_polygons;
//
//	longSize vertexs_size = host_data_info_.numberOfVertexs * sizeof(Vertex3D);
//	longSize normals_size = host_data_info_.numberOfNormals * sizeof(Normal3D);
//	longSize polygons_size = host_data_info_.numberOfPolygons * sizeof(Polygon3D);
//
//	cudaMalloc((void **)&device_vertexs, vertexs_size);
//	cudaMalloc((void**)&device_normals, normals_size);
//	cudaMalloc((void**)&device_polygons, polygons_size);
//
//	cudaMemcpy(device_vertexs, host_data_info_.allVertexs, vertexs_size, cudaMemcpyHostToDevice);
//	cudaMemcpy(device_normals, host_data_info_.allNormals, normals_size, cudaMemcpyHostToDevice);
//	cudaMemcpy(device_polygons, host_data_info_.allPolygons, polygons_size, cudaMemcpyHostToDevice);
//
//	this->device_data_info_.deviceVertexs = device_vertexs;
//	this->device_data_info_.deviceNormals = device_normals;
//	this->device_data_info_.devicePolygons = device_polygons;
//
//}
//
//void MemoryManagerGPU::AllocateMemoryForBuffer()
//{
//
//	cudaMalloc((void**)& this->device_display_buffer_, this->display_buffer_size_);
//
//}


