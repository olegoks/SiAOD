#include "hip/hip_runtime.h"

#include "GraphicEngineCUDA.cuh"
#include "__graphic_engine_/GraphicEngine.h"

#include<iostream>
void GraphicEngine::AllocateVertex2D() {

	unsigned int vertexs2d_size = data_info_.numberOfVertexs * sizeof(Vertex2D);

	hipMalloc((void**)& device_vertexs_2d_, vertexs2d_size);
	hipHostMalloc((void**)& host_vertexs_2d_, vertexs2d_size);

};


__global__ void ProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

		int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (threadIndex < number_of_vertexs) {

		//Offset
	
		float offset_x = vertexs_3d[threadIndex].x - cameraPosition.x;
		float offset_y = vertexs_3d[threadIndex].y - cameraPosition.y;
		float offset_z = vertexs_3d[threadIndex].z - cameraPosition.z;

		float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;
		
		vertexs_2d[threadIndex].x = ( k * new_x / (new_z + k) ) ;
		vertexs_2d[threadIndex].y = ( k * new_y / (new_z + k) ) ;

	}

}

__global__ void ConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {
	
	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_vertexs) {

		vertexs_2d[threadIndex].x = vertexs_2d[threadIndex].x * 1000.0f + display_width / 2;
		vertexs_2d[threadIndex].y = display_height / 2 - vertexs_2d[threadIndex].y * 1000.0f;
		
	}

}
__global__ void DrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const Normal3D* normals, const unsigned int number_of_polygons, RgbPixel* const display_buffer, const unsigned int display_width, const unsigned int display_height, Vector3D camera_dir) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_polygons * 3) {

		const unsigned int polygon_number = threadIndex / 3;
		const unsigned int vertex_number = threadIndex % 3;
		
		Normal3D normal = normals[ device_polygons[polygon_number].ratios[vertex_number].normalNumber];
		float scalar = camera_dir.x * normal.x + camera_dir.y * normal.y + camera_dir.z * normal.z;
		//if (scalar <= 0.0f) 
		{

			const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertexNumber;
			const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
			const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertexNumber;

			int x1 = vertexs_2d[first_vertex_index].x;
			int y1 = vertexs_2d[first_vertex_index].y;
			const int x2 = vertexs_2d[second_vertex_index].x;
			const int y2 = vertexs_2d[second_vertex_index].y;

			const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

			if (coordinats_are_correct) {

				const int deltaX = abs(x2 - x1);
				const int deltaY = abs(y2 - y1);
				const int signX = x1 < x2 ? 1 : -1;
				const int signY = y1 < y2 ? 1 : -1;
				//
				int error = deltaX - deltaY;

				display_buffer[display_width * y2 + x2].rgb_reserved = 0;
				display_buffer[display_width * y2 + x2].rgb_red = 255;
				display_buffer[display_width * y2 + x2].rgb_green = 0;
				display_buffer[display_width * y2 + x2].rgb_blue = 0;

				while (x1 != x2 || y1 != y2)
				{

					display_buffer[display_width * y1 + x1].rgb_reserved = 0;
					display_buffer[display_width * y1 + x1].rgb_red = 255;
					display_buffer[display_width * y1 + x1].rgb_green = 0;
					display_buffer[display_width * y1 + x1].rgb_blue = 0;

					const int error2 = error * 2;
					//
					if (error2 > -deltaY)
					{
						error -= deltaY;
						x1 += signX;
					}

					if (error2 < deltaX)
					{
						error += deltaX;
						y1 += signY;
					}

				}
			}
		}
	}

}
__global__ void  SetScreenColor(RgbPixel* const device_display_buffer, const RgbPixel rgb_pixel, const unsigned int number_of_pixels) {

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index < number_of_pixels) device_display_buffer[thread_index] = rgb_pixel;

}

void GraphicEngine::SetDisplayBufferColor(const RgbColor& rgb_color)
{
	const RgbPixel rgb_pixel = { rgb_color.rgb_blue, rgb_color.rgb_green, rgb_color.rgb_red, rgb_color.rgb_reserved };

	const unsigned int number_of_threads = 1024;

	const unsigned int number_of_blocks = (1920 * 1080 * sizeof(RgbPixel) + number_of_threads - 1) / number_of_threads;

	SetScreenColor <<< number_of_blocks, number_of_threads >>> (this->device_display_buffer_, rgb_pixel, 1920 * 1080);


}
void CPUCountingProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

	

	for (size_t i = 0; i < number_of_vertexs; i++)
	{



		//Offset
		const float offset_x = vertexs_3d[i].x - cameraPosition.x;
		const float offset_y = vertexs_3d[i].y - cameraPosition.y;
		const float offset_z = vertexs_3d[i].z - cameraPosition.z;

		const float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		const float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		const float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;

		vertexs_2d[i].x = (k * new_x / (new_z + k));
		vertexs_2d[i].y = (k * new_y / (new_z + k));
	}
	
}
void CPUCountingConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {

	

	for (size_t i = 0; i < number_of_vertexs; i++)
	{

		vertexs_2d[i].x = vertexs_2d[i].x * (float) 1000 + display_width / 2;
		vertexs_2d[i].y = display_height / 2 - vertexs_2d[i].y * 1000;

	}
}
void CPUCountingDrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const unsigned int number_of_polygons, RgbPixel* const display_buffer, const unsigned int display_width, const unsigned int display_height) {

	{
		const unsigned int number_of_lines = number_of_polygons * 3;

		for (size_t i = 0; i < number_of_lines; i++)
		{

			const unsigned int polygon_number = i / 3;
			const unsigned int vertex_number = i % 3;

			const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertexNumber;
			const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
			const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertexNumber;


			int x1 = vertexs_2d[first_vertex_index].x;
			int y1 = vertexs_2d[first_vertex_index].y;
			int x2 = vertexs_2d[second_vertex_index].x;
			int y2 = vertexs_2d[second_vertex_index].y;

			const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

			if (coordinats_are_correct) {

				const int deltaX = abs(x2 - x1);
				const int deltaY = abs(y2 - y1);
				const int signX = x1 < x2 ? 1 : -1;
				const int signY = y1 < y2 ? 1 : -1;
				//
				int error = deltaX - deltaY;

				display_buffer[display_width * y2 + x2].rgb_reserved = 0;
				display_buffer[display_width * y2 + x2].rgb_red = 255;
				display_buffer[display_width * y2 + x2].rgb_green = 0;
				display_buffer[display_width * y2 + x2].rgb_blue = 0;

				/*display_buffer[display_width * y1 + x1].rgb_reserved = 0;
				display_buffer[display_width * y1 + x1].rgb_red = 255;
				display_buffer[display_width * y1 + x1].rgb_green = 0;
				display_buffer[display_width * y1 + x1].rgb_blue = 0;*/


				while (x1 != x2 || y1 != y2)
				{
					display_buffer[display_width * y1 + x1].rgb_reserved = 0;
					display_buffer[display_width * y1 + x1].rgb_red = 255;
					display_buffer[display_width * y1 + x1].rgb_green = 0;
					display_buffer[display_width * y1 + x1].rgb_blue = 0;

					const int error2 = error * 2;
					//
					if (error2 > -deltaY)
					{
						error -= deltaY;
						x1 += signX;
					}
					if (error2 < deltaX)
					{
						error += deltaX;
						y1 += signY;
					}
				}

			}
		}
	}
}
__global__ void draw(RgbPixel* display_buffer) {
	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	display_buffer[thread_id].rgb_green = 255;

}

CameraInfo GraphicEngine::GetCameraInfo() {
	CameraInfo info;
	info.camera_pos = (*camera_->GetPosition());
	info.dis_proj_plane = camera_->GetDistanceToProjPlane();
	info.vector_x = camera_->GetVectorX();
	info.vector_y = camera_->GetVectorY();
	info.vector_z = camera_->GetDirection();
	return info;
}
void GraphicEngine::TestFunction()
{


}

void GraphicEngine::CreateMeshFrame() {

	const Vertex3D const  camera_position  = *(camera_->GetPosition());
	
	const Vector3D vector_x = camera_->GetVectorX();
	const Vector3D vector_y = camera_->GetVectorY();
	const Vector3D vector_z = camera_->GetDirection();

	const float distance_to_projection_plane = camera_->GetDistanceToProjPlane();

	const Vertex3D* const device_vertexs_3d = device_data_.deviceVertexs;
	Vertex2D* const device_vertexs_2d = device_vertexs_2d_;
	const Polygon3D* const device_polygons = device_data_.devicePolygons;
	Normal3D* device_normals = device_data_.deviceNormals;
	
	RgbColor color;
	color.rgb_blue = 20;
	color.rgb_green = 255;
	color.rgb_red = 0;

	const unsigned int number_of_threads = 1024;
	unsigned int number_of_blocks = (data_info_.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	SetDisplayBufferColor(color);

	ProjectVertexs <<<number_of_blocks, number_of_threads>>> (device_vertexs_3d, device_vertexs_2d, data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);

	ConvertInDisplayCoordinats <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, data_info_.numberOfVertexs, display_width_, display_height_);

	number_of_blocks = (data_info_.numberOfPolygons * 3 + number_of_threads - 1) / number_of_threads;

	DrawLines <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, device_polygons, device_normals, data_info_.numberOfPolygons, device_display_buffer_, display_width_, display_height_,vector_z);	

	//hipMemcpy((void**)host_display_buffer_, device_display_buffer_, display_buffer_size_, hipMemcpyDeviceToHost);
	//const Vertex3D* const host_vertexs_3d = data_info_.allVertexs;
	//hipMemcpy(host_display_buffer_, device_display_buffer_, size_of_display_buffer_, hipMemcpyDeviceToHost);
	//CPUCountingProjectVertexs(host_vertexs_3d, this->host_vertexs_2d_, this->data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);
	//CPUCountingConvertInDisplayCoordinats(host_vertexs_2d_, this->data_info_.numberOfVertexs, display_width_, display_height_);
	//Polygon3D* host_polygons = this->data_info_.allPolygons;
	//CPUCountingDrawLines(host_vertexs_2d_, host_polygons, this->data_info_.numberOfPolygons, this->host_display_buffer_, display_width_, display_height_);
	//hipMemcpy(host_display_buffer_, device_display_buffer_, size_of_display_buffer_, hipMemcpyDeviceToHost);

}


struct Proj_vertex {

	float x;
	float y;
	float _z;

};
inline __device__ void swap(Proj_vertex& a, Proj_vertex& b) {

	Proj_vertex temporary = b;
	b = a;
	a = temporary;

}
inline __device__ bool InPositiveHalfPlane(const Vertex2D& pixel, const Vertex2D& triangle_vertex, Vector2D& _normal) {

	Vector2D pixel_vector;
	pixel_vector.x = pixel.x - triangle_vertex.x;
	pixel_vector.y = pixel.y - triangle_vertex.y;


	Vector2D normal = _normal;
	float length_n = sqrt(normal.x * normal.x + normal.y * normal.y);
	normal.x /= length_n;
	normal.y /= length_n;

	float length_p = sqrt(pixel_vector.x * pixel_vector.x +  pixel_vector.y * pixel_vector.y);
	pixel_vector.x /= length_p;
	pixel_vector.y /=length_p;
	float scalar = pixel_vector.x * normal.x + pixel_vector.y * normal.y;

	if (scalar >= 0.0f) return true;
	else 
		return false;

}

struct InfoForPainting {
	Vertex3D* d_vertexs;
	Polygon3D* d_polygons;
	unsigned int number_of_polygons;
	RgbPixel* d_rgb;
	unsigned int number_of_colors;

};

__global__ void DrawPolygons(z_mutex* z_buffer,RgbPixel* display_buffer, Vertex2D* vertexs_2d, InfoForPainting info) {

	int thread_index = threadIdx.x + blockDim.x * blockIdx.x;

	if (thread_index < info.number_of_polygons) {
		//printf("%d", thread_index);
		Polygon3D polygon = info.d_polygons[thread_index];

		Proj_vertex proj_vertexs[3];
		for (int i = 0; i < 3; i++)
		{

			proj_vertexs[i].x = vertexs_2d[polygon.ratios[i].vertexNumber].x;
			//printf("x: %2f", proj_vertexs[i].x);
			proj_vertexs[i].y = vertexs_2d[polygon.ratios[i].vertexNumber].y;
			//printf("y: %2f \n", proj_vertexs[i].y);
			proj_vertexs[i]._z = 1.0f / info.d_vertexs[polygon.ratios[i].vertexNumber].z;
			
		}

		int min_x = 10000, min_y = 10000, max_x = -1, max_y = -1;

		for (int i = 0; i < 3; i++)
		{
			if (proj_vertexs[i].x < min_x) min_x = floor(proj_vertexs[i].x);
			if (proj_vertexs[i].y < min_y) min_y = floor(proj_vertexs[i].y);
			if (proj_vertexs[i].x > max_x) max_x = ceil(proj_vertexs[i].x);
			if (proj_vertexs[i].y > max_y) max_y = ceil(proj_vertexs[i].y);
		}
		
		RgbPixel polygon_color = info.d_rgb[thread_index /2];
		/*polygon_color.rgb_blue = 0;
		polygon_color.rgb_green = 0;
		polygon_color.rgb_red = 255;*/
		
		//Sorting vertexs by y 2d coordinat

		//Clockwise direction
		Vertex2D AToB;
		AToB.x = proj_vertexs[1].x - proj_vertexs[0].x;
		AToB.y = proj_vertexs[1].y - proj_vertexs[0].y;
		Vertex2D BToC;
		BToC.x = proj_vertexs[2].x - proj_vertexs[1].x;
		BToC.y = proj_vertexs[2].y - proj_vertexs[1].y;

		float crossz = AToB.x * BToC.y - AToB.y * BToC.x;
		if (crossz > 0.0f)
		{
			Proj_vertex temporary = proj_vertexs[2];
			proj_vertexs[2] = proj_vertexs[1];
			proj_vertexs[1] = temporary;
		}
	
		//if (proj_vertexs[0].x > proj_vertexs[1].x) swap(proj_vertexs[0], proj_vertexs[1]);
		//if (proj_vertexs[0].x > proj_vertexs[2].x) swap(proj_vertexs[0], proj_vertexs[2]);
		//if (proj_vertexs[1].x > proj_vertexs[2].x) swap(proj_vertexs[1], proj_vertexs[2]);
		
		float length;
		Vector2D bot_mid = { proj_vertexs[1].y - proj_vertexs[0].y, -proj_vertexs[1].x + proj_vertexs[0].x };
		length = sqrt(bot_mid.x * bot_mid.x + bot_mid.y * bot_mid.y);
		bot_mid.x /= length;
		bot_mid.y /= length;

		Vector2D mid_top = {proj_vertexs[2].y - proj_vertexs[1].y,  -proj_vertexs[2].x + proj_vertexs[1].x };
		length = sqrt(mid_top.x * mid_top.x + mid_top.y * mid_top.y);
		mid_top.x /= length;
		mid_top.y /= length;

		Vector2D top_bot = { proj_vertexs[0].y - proj_vertexs[2].y, -proj_vertexs[0].x + proj_vertexs[2].x, };
		length = sqrt(top_bot.x * top_bot.x + top_bot.y * top_bot.y);
		top_bot.x /= length;
		top_bot.y /= length;

		const Vertex2D bot = { proj_vertexs[0].x, proj_vertexs[0].y };
		const Vertex2D mid = { proj_vertexs[1].x, proj_vertexs[1].y };
		const Vertex2D top = { proj_vertexs[2].x, proj_vertexs[2].y };

		//printf("bot: %2f, %2f, mid: %2f %2f, top: %2f %2f \n", bot.x, bot.y,mid.x, mid.y,top.x, top.y);
		//printf("bot_mid: %2f %2f, mid_top: %2f %2f, top_bot: %2f %2f \n", bot_mid.x, bot_mid.y, mid_top.x, mid_top.y, top_bot.x, top_bot.y);
		for (int y = min_y; y < max_y; y++)
			for (int x = min_x; x < max_x; x++)
			{
				Vertex2D pixel;
				pixel.x = ((float)x + 0.5f);
				pixel.y = ((float)y + 0.5f);
				
				bool PixelInTriangle = InPositiveHalfPlane(pixel, bot, bot_mid) && InPositiveHalfPlane(pixel, mid, mid_top) && InPositiveHalfPlane(pixel, top, top_bot);

				if (PixelInTriangle) {
						
								
					//Vertex3D bot, mid, top;

					while ( (z_buffer + 1920 * y + x)->mutex == true ) continue;
					(z_buffer + 1920 * y + x)->mutex = true;

					//if (1.0f / (z_buffer + 1920 * y + x)->z > _z) {
					* (display_buffer + 1920 * y + x) = polygon_color;
					//}

					(z_buffer + 1920 * y + x)->mutex = false;
				}
	
				
			}
	}
	
}

void GraphicEngine::CreateFlatFrame() {

	const Vertex3D const  camera_position = *(camera_->GetPosition());

	const Vector3D vector_x = camera_->GetVectorX();
	const Vector3D vector_y = camera_->GetVectorY();
	const Vector3D vector_z = camera_->GetDirection();

	const float distance_to_projection_plane = camera_->GetDistanceToProjPlane();

	Vertex3D* const device_vertexs_3d = device_data_.deviceVertexs;
	Vertex2D* const device_vertexs_2d = device_vertexs_2d_;
	Polygon3D* const device_polygons = device_data_.devicePolygons;

	RgbColor color;
	color.rgb_blue = 20;
	color.rgb_green = 255;
	color.rgb_red = 0;

	hipMemset(z_mutex_, 0, display_width_ * display_height_ * sizeof(z_mutex));

	const unsigned int number_of_threads = 1024;
	unsigned int number_of_blocks = (data_info_.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	SetDisplayBufferColor(color);

	ProjectVertexs <<<number_of_blocks, number_of_threads >>> (device_vertexs_3d, device_vertexs_2d, data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);

	ConvertInDisplayCoordinats <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, data_info_.numberOfVertexs, display_width_, display_height_);

	number_of_blocks = (data_info_.numberOfPolygons * threads_per_triangle_ + number_of_threads - 1) / number_of_threads;


	InfoForPainting info = { device_vertexs_3d, device_polygons, data_info_.numberOfPolygons, (RgbPixel*)device_data_.device_colors, data_info_.numberOfRgbColors };
	printf("%d", data_info_.numberOfRgbColors);
	DrawPolygons <<< number_of_blocks, number_of_threads >> > (z_mutex_, device_display_buffer_, device_vertexs_2d, info);

	//system("pause");

}



