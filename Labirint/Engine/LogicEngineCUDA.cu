#include "hip/hip_runtime.h"
#include "__engine_/__logic_engine_/LogicEngine.h"
static const float pi = 3.14159265;
__global__ void RotateModelCuda(Vertex3D* const device_vertexs, const unsigned int number_of_vertexs, const float sin_alpha, const float cos_alpha, const Vector3D around_vector, const Vertex3D around_vertex) {

	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_id < number_of_vertexs) {

		float VX = around_vector.x;
		float VY = around_vector.y;
		float VZ = around_vector.z;

		float XP = device_vertexs[thread_id].x - around_vertex.x;
		float YP = device_vertexs[thread_id].y - around_vertex.y;
		float ZP = device_vertexs[thread_id].z - around_vertex.z;

		device_vertexs[thread_id].x = (cos_alpha + (1 - cos_alpha) * VX * VX) * XP + ((1 - cos_alpha) * VX * VY - sin_alpha * VZ) * YP + ((1 - cos_alpha) * VX * VZ + sin_alpha * VY) * ZP + around_vertex.x;
		device_vertexs[thread_id].y = ((1 - cos_alpha) * VY * VX + sin_alpha * VZ) * XP + (cos_alpha + (1 - cos_alpha) * VY * VY) * YP + ((1 - cos_alpha) * VY * VZ - sin_alpha * VX) * ZP + around_vertex.y;
		device_vertexs[thread_id].z = ((1 - cos_alpha) * VZ * VX - sin_alpha * VY) * XP + ((1 - cos_alpha) * VZ * VY + sin_alpha * VX) * YP + (cos_alpha + (1 - cos_alpha) * VZ * VZ) * ZP + around_vertex.z;

	}

}

void LogicEngine::RotateModel(unsigned int model_id, const float alpha_degree, const Vector3D& around_vector, const Vertex3D& around_point) {

	const ModelInfo3D current_model = data_info_.allModels[model_id];

	float alpha_rad = alpha_degree * pi / 180.0f;

	const float sin_alpha = sin(alpha_rad);
	const float cos_alpha = cos(alpha_rad);

	const unsigned int number_of_threads = 1024;
	const unsigned int number_of_blocks = (current_model.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	RotateModelCuda <<<number_of_blocks, number_of_threads >>> (current_model.device_vertexs_pointer, current_model.numberOfVertexs, sin_alpha, cos_alpha, around_vector, around_point);

}

__global__ void ScaleModelCuda(Vertex3D* const vertexs, const unsigned int number_of_vertexs, const float coefficient) {

	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_id < number_of_vertexs) {

		vertexs[thread_id].x *= coefficient;
		vertexs[thread_id].y *= coefficient;
		vertexs[thread_id].z *= coefficient;

	}

}

void LogicEngine::ScaleModel(const unsigned int model_id, const float coefficient) {

	const ModelInfo3D current_model = data_info_.allModels[model_id];
	
	const unsigned int number_of_threads = 1024;
	const unsigned int number_of_blocks = (current_model.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	ScaleModelCuda <<< number_of_blocks, number_of_threads >>> (current_model.device_vertexs_pointer, current_model.numberOfVertexs, coefficient);

}

__global__ void TranslateModelCuda(Vertex3D* const vertexs, const unsigned int number_of_vertexs, const Vertex3D translate_vertex) {

	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_id < number_of_vertexs) {

		vertexs[thread_id].x += translate_vertex.x;
		vertexs[thread_id].y += translate_vertex.y;
		vertexs[thread_id].z += translate_vertex.z;

	}
}

void LogicEngine::TranslateModel(const unsigned int model_id, const Vertex3D& translate_vertex)
{
	const ModelInfo3D current_model = data_info_.allModels[model_id];

	const unsigned int number_of_threads = 1024;
	const unsigned int number_of_blocks = (current_model.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	TranslateModelCuda <<<  number_of_blocks, number_of_threads >>> (current_model.device_vertexs_pointer, current_model.numberOfVertexs, translate_vertex);

}
